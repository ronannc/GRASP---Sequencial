#include "hip/hip_runtime.h"
/*Bin Packing - two Dimensional
Problema da mochila com duas dimensoes.
Tal problema consiste em escolher o maior numero de itens que comportem em uma mochila (compartimento)

O dataset osado foi o presente na biblioteca ORLIB (OR-Library)

Cada arquivo contem:
Numero de problemas teste
Identificador do Problema
Capacidade do compartimento, numero de itens, numero de itens presentes na melhor solu��o conhecida.
Tamanho dos itens.
*/

#include "objeto.h"
#include "SimpleGRASP.h"
#include "EntradaDeDados.h"
#include "Selection_Sort.h"
#include "time.h"

#include "SaidaDados.h"

int maximo(int a, int b);
int knapSack(int W, item *itens, int n);

int main() {
	
	//semente para gerador de numeros aleatorios
	srand(time(NULL));
	int seed = rand();
	
	//temperatura inicial e o decaimento da temperatura para SA
	int temperatura = 100;
	int decaimento_temperatura = 1;

	//tamanha usado para tornar a gera��o da solu��o inicial e busca aleatoria, se 1 fica modo guloso
	int tamanho_RCL = 10;
	
	//numero de itera��es que o GRASP fara
	int max_iter = 1000;

	// numero de elementos
	int quantidade_itens = 0;

	// capacidade mochia
	int capacidade_mochila = 0;

	//recebendo entrada, quantidade e capacidade da mochila
	entrada_dados(quantidade_itens, capacidade_mochila);

	//Vetor usado para guardar peso, valor e ganho
	item *itens;
	itens = (item *)malloc(quantidade_itens * sizeof(item));

	//verificando se alocou o vetor
	if (!itens) {
		printf("Sem memoria disponivel! (itens)\n");
		exit(1);
	}

	//recebendo peso e valor de cada item
	entrada_dados_vetor(itens, quantidade_itens);

	//ordenando os itens com rela��o ao valor/peso
	selection_sort(itens, quantidade_itens);

	//imprime os itens e seu indece de ganho de modo ordenado
	
	//for (int i = 0; i < quantidade_itens; i++) {
	//	printf("%d %d %f\n", itens[i].peso, itens[i].valor, itens[i].ganho);
	//}

	//system("pause");

	printf("====== Bin Packing - Bi Dimensional ======\n");
	printf("\n");
	printf("              IFMG - Formiga            \n");
	printf(" Desenvolvido por: Ronan Nunes Campos   \n");
	printf(" Matricula: 0011919                     \n");
	printf("\n");
	printf(" Dados do problema           \n\n");
	printf(" Numero de Itens: %d                    \n", quantidade_itens);
	printf(" Capacidade Mochila: %d                 \n", capacidade_mochila);
	printf(" Numero de Itera��es: %d                \n", max_iter);
	printf(" Geradas %d soluc�es\n", max_iter);
	printf(" Rodando na CPU                          \n");
	printf("===========================================\n");

	//vetor para a solu��o do problema. se 1 esta na mochila de 0 esta fora
	bool *soluctions;
	soluctions = (bool *)malloc(quantidade_itens * sizeof(bool));

	//verificando se alocou o vetor
	if (!soluctions) {
		printf("Sem memoria disponivel! (soluctions)\n");
		exit(1);
	}

	//iniciando a solu��o com todos os itens fora da mochila == 0
	for (int i = 0; i < quantidade_itens; i++) {
		soluctions[i] = 0;
	}

	// Rodando GRASP sequencial recursivo e calculando o tempo de execu��o.
	clock_t t0, tf;
	double tempo_gasto;

	//t0 = clock();
		//printf("recursivo: %d\n", knapSack(capacidade_mochila, itens, quantidade_itens));
	//tf = clock();
	
	//tempo gasto pelo GRASP
	//tempo_gasto = ((double)(tf - t0)) / CLOCKS_PER_SEC;
	//printf("\n");
	//printf("Tempo gasto recursivo: %lf s\n", tempo_gasto);

	//rodadndo o GRASP sequncial
	//valor da solu��o
	int max_valor = 0;
	saida_header();
	for (int k = 0; k < 100; k++) {
	t0 = clock();
		simpleGRASP(max_iter, quantidade_itens, capacidade_mochila, itens, soluctions, temperatura, decaimento_temperatura, tamanho_RCL, seed, max_valor);
	tf = clock();

	tempo_gasto = ((double)(tf - t0)) / CLOCKS_PER_SEC;
	saida_body(max_valor, tempo_gasto, max_iter);
	printf("\n");
	printf("Tempo gasto GRASP: %lf s\n", tempo_gasto);

	printf("===========================================\n\n");
	printf("max valor: %d\n", max_valor);
	for (int i = 0; i < quantidade_itens; i++) {
		printf("%d ", soluctions[i]);
	}

	printf("\n");
	printf("fim :)\n\n");
	}
	free(itens);
	free(soluctions);
	system("pause");
	return 0;
}
// A utility function that returns maximum of two integers
int maximo(int a, int b) {
	if (a > b) {
		return a;
	}
	else {
		return b;
	}
}

// Returns the maximum value that can be put in a knapsack of capacity W
int knapSack(int W, item *itens, int n)
{
	// Base Case
	if (n == 0 || W == 0)
		return 0;

	// If weight of the nth item is more than Knapsack capacity W, then
	// this item cannot be included in the optimal solution
	if (itens[n - 1].peso > W)
		return knapSack(W, itens, n - 1);

	// Return the maximum of two cases: 
	// (1) nth item included 
	// (2) not included
	else return maximo(itens[n - 1].valor + knapSack(W - itens[n - 1].peso, itens, n - 1),
		knapSack(W, itens, n - 1)
	);
}