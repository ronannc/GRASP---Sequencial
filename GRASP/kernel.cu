#include "hip/hip_runtime.h"
#include "SimplleGRASP.h"

// numero de elementos
int size = 0;
// numero maximo de itera�oes
int maxIter = 0;
// numero capacidade mochia
int capacidade = 0;

int main(){
	
	node *vectorPeso;
	vectorPeso = (node *)malloc(sizeof(node));

	if (!vectorPeso) {
		printf("Sem memoria disponivel!\n");
		exit(1);
	}

	node *vectorValor;
	vectorValor = (node *)malloc(sizeof(node));

	if (!vectorValor) {
		printf("Sem memoria disponivel!\n");
		exit(1);
	}

	inicia(vectorPeso);
	inicia(vectorValor);

	FILE *arq;
	int aux = 0;
	char s;
	arq = fopen("entrada.txt", "r");

	if (arq == NULL) {

		printf("Erro, nao foi possivel abrir o arquivo\n");
	}else {

		fscanf(arq, "%c %d\n", &s, &size);
		fscanf(arq, "%c %d\n", &s, &capacidade);
		fscanf(arq, "%c %d\n", &s, &maxIter);
		fscanf(arq, "%c\n", &s);
		
		for (int i = 0; i < size; i++) {
			fscanf(arq, "%d\n", &aux);
			insereInicio(vectorPeso, aux);
		}
		
		fscanf(arq, "%c\n", &s);
		//printf("%c", s);
		for (int i = 0; i < size; i++) {
			fscanf(arq, "%d\n", &aux);
			insereInicio(vectorValor, aux);
		}
	}
			
	fclose(arq);

	printf("==========================================\n");
	printf("Dados do problema:\n");
	printf("Capacidade da mochila: %d\n", capacidade);
	printf("Numero de itens: %d\n", size);
	printf("Numero de itera��es: %d\n", maxIter);
		
	clock_t t0, tf;
	double tempo_gasto;
	
	t0 = clock();
		simplleGrasp(maxIter, size, capacidade, vectorPeso, vectorValor);
	tf = clock();
	
	tempo_gasto = ((double)(tf - t0)) / CLOCKS_PER_SEC;
	
	printf("Tempo gasto: %lf s\n", tempo_gasto);

	printf("===========================================\n\n");
	
	free(vectorPeso);
	free(vectorValor);
	system("pause");
    return 0;
}